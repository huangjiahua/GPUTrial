#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

using namespace std;

__global__
void
add_each(int *buf, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int *ptr;

    hipMalloc((void**)&ptr, sizeof(int));
    
    while (tid < n) {
        (*ptr) = buf[tid];
        (*ptr) += 1;
        buf[tid] = *ptr;
        tid += stride;
    }
    hipFree(ptr);
}

int main() {
    int nums[100];
    int *dev_nums;
    for (int i = 0; i < 100; i++)
        nums[i] = i;
    
    hipMalloc((void**)&dev_nums, sizeof(int) * 100);
    hipMemcpy(dev_nums, nums, sizeof(int) * 100, hipMemcpyHostToDevice);
    add_each<<<4, 16>>>(dev_nums, 100);
    for (int &i : nums) nums[i] = 0;
    hipMemcpy(nums, dev_nums, sizeof(int) * 100, hipMemcpyDeviceToHost);
    for (int i : nums) cout << i << endl;
    hipFree(dev_nums);
}